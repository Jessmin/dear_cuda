#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"

void sumArrays(float * a,float * b,float * res,const int size)
{
  for(int i=0;i<size;i+=4)
  {
    res[i]=a[i]+b[i];
    res[i+1]=a[i+1]+b[i+1];
    res[i+2]=a[i+2]+b[i+2];
    res[i+3]=a[i+3]+b[i+3];
  }
}

__global__ void sumArraysGPU(float *a,float *b, float *res){
  int i = threadIdx.x;
  res[i]=a[i]+b[i];
}

int main(int argc,char **argv){
  int dev = 0;
  hipSetDevice(dev);

  int nElem = 32;
  printf("Vector Size:%d\n",nElem);

  int nByte = sizeof(float) * nElem;
  



}